#include "hip/hip_runtime.h"
//
// Created by 92571 on 2024/7/2.
//
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "common.h"
// ----------------------------------------------------------------------------
// CPU code reference

void adamw_cpu(float* params_memory, const float* grads_memory, float* m_memory, float* v_memory, int t, long num_parameters, float learning_rate=1e-3, float beta1=0.9, float beta2=0.999, float eps=1e-8, float weight_decay=0.0) {

    for (int i = 0; i < num_parameters; i++) {
        float param = params_memory[i];
        float grad = grads_memory[i];

        float m = beta1 * m_memory[i] + (1.0f - beta1) * grad;
        float v = beta2 * v_memory[i] + (1.0f - beta2) * grad * grad;

        float m_hat = m / (1.0f - powf(beta1, t));
        float v_hat = v / (1.0f - powf(beta2, t));

        m_memory[i] = m;
        v_memory[i] = v;
        params_memory[i] -= learning_rate * (m_hat / (sqrtf(v_hat) + eps) + weight_decay * param);
    }
}

__global__ void adamw_kernel1(float* params_memory, const float* grads_memory, float* m_memory, float* v_memory, long num_parameters,
                              float learning_rate, float beta1, float beta2, float beta1_correction, float beta2_correction, float eps, float weight_decay) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_parameters) return;
    m_memory[i] = beta1 * m_memory[i] + (1.0f - beta1) * grads_memory[i];
    v_memory[i] = beta2 * v_memory[i] + (1.0f - beta2) * grads_memory[i] * grads_memory[i];
    float m_hat = m_memory[i] / beta1_correction;
    float v_hat = v_memory[i] / beta2_correction;
    params_memory[i] -= learning_rate * (m_hat / (sqrtf(v_hat) + eps) + weight_decay * params_memory[i]);
}

